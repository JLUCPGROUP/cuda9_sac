
#include <hip/hip_runtime.h>
////#pragma once
//// CUDA Runtime
////#include <cuda_runtime.h>
////#include <device_functions.h>
////#include <device_launch_parameters.h>
//// Utilities and system includes
////#include <helper_cuda.h>
////#include <helper_functions.h>
////#include <thrust/device_vector.h>
////#include <thrust/scan.h>
////#include <iostream>
////#include <math.h>
////#include "XModel.h"
////#include "math_functions.hpp"
////#include "cuda_runtime_api.h"
////
////using namespace cudacp;
////
////#ifndef MIN
////#define MIN(x,y) ((x < y) ? x : y)
////#endif
////// num_threads
////static const int num_threads = 128;
////static const int U32_SIZE = sizeof(u32); ///<4
////static const int U32_BIT = U32_SIZE * 8;	///<32
////static const int U32_POS = 5;
////static const int U32_MOD_MASK = 31;
////
////struct int_predicate
////{
////	__host__ __device__	bool operator()(const int x)
////	{
////		return x > 0;
////	}
////};
////
////// 一个bitDom[x]的长度
////__constant__ int D_BITDOM_INTSIZE;
////// 整个bitDom的长度
////
////__constant__ int D_BITDOMS_INTSIZE;
////
////// bit支持， uint2 bitSup[c][a][idx].x = bitSup[c,x,a,idx]
////// bit支持， uint2 bitSup[c][a][idx].y = bitSup[c,y,a,idx]
//////__device__ uint2*** d_bitSup;
//////__host__ uint2*** h_bitSup;
//////__device__ u32** d_bitDom;
//////__host__ u32** h_bitDom;
//////__device__ u32**
////__constant__ int D_NUM_BD_BLOCK;
////__constant__ int D_NUM_CS_SIZE_BLOCKS;
//////////////////////////////////////////////////////////////////////////////
//////	一些GPU常量
//////////////////////////////////////////////////////////////////////////////
////__device__ __managed__ int NUM_BD_BLOCK;
////__device__ __managed__ int NUM_CS_SIZE_BLOCKS;
////// 一个变量论域的int长度
////__device__ __managed__ int BITDOM_INTSIZE;
////// 整个变量集合的论域的int长度 
////__device__ __managed__ int BITDOMS_INTSIZE;
////// 一个约束的bitsup的int长度 
////__device__ __managed__ int BITSUP_INTSIZE;
////// 整个约束集合的bitsup的int长度
////__device__ __managed__ int BITSUPS_INTSIZE;
//////子问题论域总长度
////__device__ __managed__ int BITSUBDOMS_INTSIZE;
//////变量个数
////__device__ __managed__ int VS_SIZE;
//////约束个数
////__device__ __managed__ int CS_SIZE;
//////主问题约束压缩BLOCK数
////__device__ __managed__ int MCC_BLOCK;
//////////////////////////////////////////////////////////////////////////////
//////	一些GPU变量
//////////////////////////////////////////////////////////////////////////////
////__device__ __managed__ int M_Qsize;
////
//////////////////////////////////////////////////////////////////////////////
//////  GPU约束记录信息，不可更改
//////////////////////////////////////////////////////////////////////////////
//////	每个变量的int大小
////__device__ __managed__ int *vars_size;
////// 存储约束的scope，类型int3，scope.x: x.id; scope.y: y.id; scope.z: c.id
////__device__ __managed__ int3* scope;
////// 最大dom
////__device__ __managed__ int MAX_DOM_SIZE;
////// subCon长度
////__device__ __managed__ int SUBCON_SIZE;
////
////
//////__device__ __managed__ int BITDOM_SIZE;
//////__device__ __managed__ int
////// 主问题数据结构，使用UM
////// 表示约束网络论域
////__device__ __managed__ u32* bitDom;
////// 表示约束，不可修改
////__device__ __managed__ uint2* bitSup;
////////类似队列，存储约束id
//////__device__ __managed__ int *mainCon;
////////子问题数据结构
//////表示子问题的约束网络论域。
////__device__ __managed__ u32* bitSubDom;
////////类似队列，存储子问题约束id subCon.x: variable，subCon.y: value，subCon.z: c.id
//////__device__ __managed__ ushort3* subCon;
//////标记主问题变量域是否删减，初始化全部为1
////__device__ __managed__ int* M_VarPre;
//////标记主问题约束是否需检查，初始化全部为1
////__device__ __managed__ int* M_ConPre;
//////主问题约束传播队列(压缩版)
////__device__ __managed__ int* M_ConEvt;
//////主问题约束传播队列
////__device__ __managed__ int* M_Con;
////
//////标记子问题变量域是否删减，初始化全部为1
////__device__ __managed__ int* S_VarPre;
//////记录子问题变量域
////__device__ __managed__ uint3* S_Var;
//////标记子问题约束是否需检查，初始化全部为1
////__device__ __managed__ int* S_ConPre;
//////子问题约束传播队列(压缩版)
////__device__ __managed__ int3* S_ConEvt;
//////子问题约束传播队列
////__device__ __managed__ int3* S_Con;
//////
////int* MCC_BlocksCount;
////int* MCC_BlocksOffset;
////
////thrust::device_vector<int> MCC_BCount;
////thrust::device_vector<int> MCC_BOffset;
//////
//////__device__ __managed__ ushort4* subVar;
////////标记子问题发生改动的变量id，初始化全部为0
//////__device__ __managed__ unsigned short* subEvtVar;
////////标记子问题发生改动的约束id，初始化全部为1
//////__device__ __managed__ int* subEvtCon;
////
////static const u32 U32_MASK1[32] = {
////	0x80000000, 0x40000000, 0x20000000, 0x10000000,
////	0x08000000, 0x04000000, 0x02000000, 0x01000000,
////	0x00800000, 0x00400000, 0x00200000, 0x00100000,
////	0x00080000, 0x00040000, 0x00020000, 0x00010000,
////	0x00008000, 0x00004000, 0x00002000, 0x00001000,
////	0x00000800, 0x00000400, 0x00000200, 0x00000100,
////	0x00000080, 0x00000040, 0x00000020, 0x00000010,
////	0x00000008, 0x00000004, 0x00000002, 0x00000001,
////};
////
////static const u32 U32_MASK0[32] = {
////	0x7FFFFFFF, 0xBFFFFFFF, 0xDFFFFFFF, 0xEFFFFFFF,
////	0xF7FFFFFF, 0xFBFFFFFF, 0xFDFFFFFF, 0xFEFFFFFF,
////	0xFF7FFFFF, 0xFFBFFFFF, 0xFFDFFFFF, 0xFFEFFFFF,
////	0xFFF7FFFF, 0xFFFBFFFF, 0xFFFDFFFF, 0xFFFEFFFF,
////	0xFFFF7FFF, 0xFFFFBFFF, 0xFFFFDFFF, 0xFFFFEFFF,
////	0xFFFFF7FF, 0xFFFFFBFF, 0xFFFFFDFF, 0xFFFFFEFF,
////	0xFFFFFF7F, 0xFFFFFFBF, 0xFFFFFFDF, 0xFFFFFFEF,
////	0xFFFFFFF7, 0xFFFFFFFB, 0xFFFFFFFD, 0xFFFFFFFE,
////};
////
//////__forceinline__ int  GetBitDomIndex(int var_id)
//////{
//////	return var_id * BITDOM_INTSIZE;
//////}
////
////// 根据x和index获得bitDom位置
////#define GetBitDomIndex(x, i) (x * BITDOM_INTSIZE + i)
////// 根据落在最后文字的值的个数获取bit表示的偏移量
////#define GetOffSet(x)(U32_BIT - (x & U32_MOD_MASK))
////
////#define GetBitSubDomStartIndex(x,a)((x * MAX_DOM_SIZE + a) * BITDOMS_INTSIZE)
////#define GetBitSubDomIndex(x, a, y, i)(GetBitSubDomStartIndex(x,a) + GetBitDomIndex(y, i))
////
////__device__ bool IsGtZero(int x)
////{
////	return x > 0;
////}
////
////__inline__ __device__ __host__ int GetTopNum(int num_elements, int num_threads)
////{
////	return (num_elements + (num_threads - 1)) / num_threads;
////}
//////************************************
////// Method:    intsizeof
////// FullName:  intsizeof
////// Access:    public 
////// Returns:   int
////// Qualifier: 获取用bit表示的int长度
////// Parameter: const int x
//////************************************
////inline int intsizeof(const int x)
////{
////	return (int)ceil((float)x / U32_BIT);
////}
////
////__device__ __inline__ int pow2i(int e)
////{
////	return 1 << e;
////}
//////__global__ void enforceACMain(u32* bitDom, u32* bitSup, u32* M_Con, u32* M_ConEvt, u32* M_ConPre, u32* M_VarPre)
//////{
//////
//////}
////
//////通过已改变变量生成约束队列
////__global__ void GenConPre(int *VarPre, int* BlocksCount, int3* scp, int len)
////{
////	const int idx = blockDim.x*blockIdx.x + threadIdx.x;
////	if (idx < len)
////	{
////		int3 sp = scp[idx];
////		int pred;
////		if (VarPre[sp.x] == 1 || VarPre[sp.y] == 1)
////			pred = 1;
////		else
////			pred = 0;
////
////		int BC = __syncthreads_count(pred);
////
////		if (threadIdx.x == 0)
////		{
////			BlocksCount[threadIdx.x] = BC;
////		}
////	}
////}
////
////__global__ void CompactQ(int *VarPre, int* ConEvt, int* BOffset, int3* scp, int len)
////{
////	int idx = threadIdx.x + blockIdx.x*blockDim.x;
////	extern __shared__ int warpTotals[];
////	一个线程块内有128个线程
////	一个块内有4个线程束
////	if (idx < len)
////	{
////		int3 sp = scp[idx];
////		int pred;
////		获得判定
////		if (VarPre[sp.x] == 1 || VarPre[sp.y] == 1)
////			pred = 1;
////		else
////			pred = 0;
////
////		warp index
////		线程束索引
////		int w_i = threadIdx.x / warpSize;
////		thread index within a warp
////		线程束内线程索引
////		int w_l = idx % warpSize;
////		thread mask (ERROR IN THE PAPERminus one is required)
////		线程掩码
////		INT_MAX = 1111 1111 1111 1111 1111 1111 1111 1111 
////		若线程内id=0，右移32-0-1 = 31位 右侧剩下1位
////		若线程内id=5，右移32-5-1 = 26位 右侧剩下6位
////		若线程内id=31，右移32-31-1 = 0位 右侧剩下32位
////		线程束内threid|  31~~~~~~0
////		ballot对应位置|   1......1
////		int t_m = INT_MAX >> (warpSize - w_l - 1);
////		balres = number whose ith bit is one if the ith's thread pred is true masked up to the current index in warp
////		线程内局部变量pred = 1，与掩码按位与但过滤掉超过该线程id的记录，只保留变量前面的判定
////		int b = __ballot(pred) & t_m;
////		popc count the number of bit one. simply count the number predicated true BEFORE MY INDEX
////		计算只计算当前线程索引对应的前N个的位数之和
////		即为线程束内排他扫描
////		int t_u = __popc(b);
////
////		由每个线程束最后一个线程写入共享内存，对应id为线程束ID，将本线程ID加回，
////		将包含求和的最终值写入共享内存，不包含求和的值没有被覆盖
////		warpTotals长度为4
////		if (w_l == warpSize - 1)
////			warpTotals[w_i] = t_u + pred;
////
////		__syncthreads();
////
////		线程束id为0，线程束内线程id，若blockDim.x = 128，则w_l < 128/32 = 4
////		线程块内第一个线程束的前（4）个线程束工作，w_l < 活动线程束数（4），即每个线程束被一个线程运行
////		if (w_i == 0 && w_l < blockDim.x / warpSize)
////		{
////			int w_i_u = 0;
////			for (int j = 0; j <= 5; ++j)
////			{
////				# of the ones in the j'th digit of the warp offsets
////				0->5 6个位置：
////				000 001
////				000 010
////				000 100
////				001 000
////				010 000
////				100 000
////				int b_j = __ballot(warpTotals[w_l] & pow2i(j));
////				w_i_u += (__popc(b_j & t_m)) << j;
////				printf("indice %i t_m=%i,j=%i,b_j=%i,w_i_u=%i\n",w_l,t_m,j,b_j,w_i_u);
////			}
////			warpTotals[w_l] = w_i_u;
////		}
////		__syncthreads();
////
////		if (pred)
////			ConEvt[t_u + warpTotals[w_i] + BOffset[blockIdx.x]] = scp[idx].z;
////
////	}
////}
////
////void CompactQueueMain()
////{
////	//以约束数量启动
////	//P1
////	GenConPre << <MCC_BLOCK, num_threads >> >(M_VarPre, MCC_BlocksCount, scope, CS_SIZE);
////	cudaDeviceSynchronize();
////	//P2
////	thrust::exclusive_scan(MCC_BCount.begin(), MCC_BCount.end(), MCC_BOffset.begin());
////	cudaDeviceSynchronize();
////	//P3
////	//每个约束一个线程进行归约,共享内存大小 = 一个块内线程束的个数,用来装载线程束计算结果
////	CompactQ << <MCC_BLOCK, num_threads, sizeof(int)*(num_threads / warpSize) >> >(M_VarPre, M_Con, MCC_BlocksOffset, scope, CS_SIZE);
////}
////
////#define GetBitSupIndexByINTPrstn(cid,x_val,y_val) (cid * BITSUP_INTSIZE + x_val * BITDOM_INTSIZE + y_val)
////
////__inline__ __device__ __host__ int2 GetBitSupIndexByTuple(int cid, int2 t)
////{
////	return make_int2(cid * BITSUP_INTSIZE + t.x * BITDOM_INTSIZE + (t.y >> U32_POS), cid * BITSUP_INTSIZE + t.y * BITDOM_INTSIZE + (t.x >> U32_POS));
////}
////
////__inline__ __device__ __host__ int GetBitSupIndexById(int cid)
////{
////	return cid * BITSUP_INTSIZE;
////}
////
////void DelGPUModel();
////
////void BuildBitModel(XModel *xm)
////{
////#pragma region 计算常量
////	BITDOM_INTSIZE = intsizeof(xm->feature.max_dom_size);
////	MAX_DOM_SIZE = xm->feature.max_dom_size;
////	VS_SIZE = xm->feature.vs_size;
////	CS_SIZE = xm->feature.cs_size;
////	BITDOMS_INTSIZE = BITDOM_INTSIZE * VS_SIZE;
////	BITSUP_INTSIZE = MAX_DOM_SIZE * BITDOM_INTSIZE;
////	BITSUPS_INTSIZE = BITSUP_INTSIZE * CS_SIZE;
////	BITSUBDOMS_INTSIZE = VS_SIZE * MAX_DOM_SIZE * BITDOMS_INTSIZE;
////	SUBCON_SIZE = VS_SIZE * MAX_DOM_SIZE * CS_SIZE;
////#pragma endregion 计算常量
////#pragma region 约束网络信息
////	//cudaMallocManaged(&vars_size, sizeof(int) * VS_SIZE);
////	//// 初始化变量域大小
////	//for (int i = 0; i < xm->feature.vs_size; ++i)
////	//{
////	//	XVar* v = xm->vars[i];
////	//	XDom* d = xm->doms[v->dom_id];
////	//	vars_size[i] = d->size;
////	//}
////
////	// 初始化scope
////	cudaMallocManaged(&scope, sizeof(int3) * CS_SIZE);
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		XCon *c = xm->cons[i];
////		scope[i].x = c->scope[0];
////		scope[i].y = c->scope[1];
////		scope[i].z = c->id;
////	}
////
////	////显示
////	//for (int i = 0; i < CS_SIZE; ++i)
////	//{
////	//	printf("scope[%d] = {%d, %d}\n", scope[i].z, scope[i].x, scope[i].y);
////	//}
////#pragma endregion 约束网络信息
////#pragma region 拷贝bitDom
////	cudaMallocManaged(&bitDom, sizeof(u32) * BITDOMS_INTSIZE);
////	cudaMallocManaged(&M_VarPre, sizeof(int) * VS_SIZE);
////
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		XVar* v = xm->vars[i];
////		XDom* d = xm->doms[v->dom_id];
////		const int dom_size = d->size;
////		// 当前变量的实际INT长度
////		const int dom_int_size = intsizeof(dom_size);
////
////		for (int j = 0; j < BITDOM_INTSIZE; ++j)
////		{
////			const int idx = GetBitDomIndex(i, j);
////			//printf("idx = %d\n", idx);
////			// 三种情况
////			if (j < dom_int_size - 1)
////				bitDom[idx] = UINT32_MAX;
////			else if (j == dom_int_size - 1)
////				bitDom[idx] = UINT32_MAX << GetOffSet(dom_size);
////			else
////				bitDom[idx] = 0;
////		}
////
////		M_VarPre[i] = 1;
////	}
////
////	//for (int i = 0; i < VS_SIZE; ++i)
////	//{
////	//	for (int j = 0; j < BITDOM_INTSIZE; ++j)
////	//	{
////	//		int idx = GetBitDomIndex(i, j);
////	//		printf("var = %d, j = %d, idx = %d, bitDom = %x, pre= %x\n", i, j, idx, bitDom[idx], M_VarPre[i]);
////	//	}
////	//}
////#pragma endregion 拷贝bitDom
////#pragma region 创建bitSubDom
////	cudaMallocManaged(&bitSubDom, sizeof(u32)*BITDOMS_INTSIZE*VS_SIZE*MAX_DOM_SIZE);
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			const int start_idx = GetBitSubDomStartIndex(i, j);
////			for (int k = 0; k < BITSUBDOMS_INTSIZE; ++k)
////				bitSubDom[start_idx + k] = bitDom[k];
////			//最后将(i,j)的bitDom 改掉
////			//获取i,j,i的起始地址，
////			const int ijistart = start_idx + i*BITDOM_INTSIZE;
////			for (int k = 0; k < BITDOM_INTSIZE; ++k)
////				// j在索引K的范围内:j/32,将第j%32位置为1
////				if (k == j >> U32_POS)
////					bitSubDom[ijistart + k] = U32_MASK1[j&U32_MOD_MASK];
////			//其它位置为0
////				else
////					bitSubDom[ijistart + k] = 0;
////		}
////	}
////
////	//for (int i = 0; i < VS_SIZE; ++i)
////	//{
////	//	for (int j = 0; j < MAX_DOM_SIZE; ++j)
////	//	{
////	//		printf("sub problem:(%d, %d): ", i, j);
////	//		const int start_idx = GetBitSubDomStartIndex(i, j);
////	//		for (int k = 0; k < BITDOMS_INTSIZE; ++k)
////	//		{
////	//			printf("%x ", bitSubDom[start_idx + k]);
////	//		}
////	//		printf("\n");
////	//	}
////	//}
////#pragma endregion 创建bitSubDom
////#pragma region 拷贝bitSup
////	cudaMallocManaged(&bitSup, sizeof(uint2) * BITSUPS_INTSIZE);
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		XCon* c = xm->cons[i];
////		XRel* r = xm->rels[c->rel_id];
////		XVar* v[2] = { xm->vars[c->scope[0]], xm->vars[c->scope[1]] };
////		XDom* d[2] = { xm->doms[v[0]->dom_id], xm->doms[v[1]->dom_id] };
////
////		//初始化位矩阵
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			for (int k = 0; k < BITDOM_INTSIZE; ++k)
////			{
////				const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
////				if (j < d[0]->size && (k < (d[1]->size >> U32_POS)))
////				{
////					//支持取0x0000..., 冲突取0xFFF...
////					bitSup[idx].x = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].y = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////				}
////				else if (k == (d[1]->size >> U32_POS))
////				{
////					bitSup[idx].x = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].y = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].x <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
////					bitSup[idx].y <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
////				}
////				else
////				{
////					bitSup[idx].x = 0;
////					bitSup[idx].y = 0;
////				}
////			}
////		}
////		//向位矩阵中填充值
////		for (int j = 0; j < r->size; ++j)
////		{
////			const int2 t = make_int2(r->tuples[j][0], r->tuples[j][1]);
////			//printf("c_id= %d, %d, %d\n", c->id, t.x, t.y);
////			const int2 idx = GetBitSupIndexByTuple(c->id, t);
////			//printf("idx = %d, %d\n", idx.x, idx.y);
////			if (r->sem == SEM_SUPPORT)
////			{
////				bitSup[idx.x].x |= U32_MASK1[t.y & U32_MOD_MASK];
////				bitSup[idx.y].y |= U32_MASK1[t.x & U32_MOD_MASK];
////			}
////			else
////			{
////				bitSup[idx.x].x &= U32_MASK0[t.y & U32_MOD_MASK];
////				bitSup[idx.y].y &= U32_MASK0[t.x & U32_MOD_MASK];
////			}
////		}
////		//// 初始化位矩阵
////		//for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		//{
////		//	printf("c_id = %d, j = %d: ", i, j);
////		//	for (int k = 0; k < BITDOM_INTSIZE; ++k)
////		//	{
////		//		const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
////		//		printf("%x, %x", bitSup[idx].x, bitSup[idx].y);
////		//	}
////		//	printf("\n");
////		//}
////	}
////#pragma endregion 拷贝bitSup
////#pragma region 生成约束
////	cudaMallocManaged(&M_Con, sizeof(int)*CS_SIZE);
////	cudaMallocManaged(&M_ConEvt, sizeof(int) * CS_SIZE);
////	cudaMallocManaged(&M_ConPre, sizeof(int)*CS_SIZE);
////
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		M_Con[i] = i;
////		M_ConEvt[i] = i;
////		M_ConPre[i] = 1;
////		//printf("i = %d , M_Con = %d, M_ConEvt = %d, M_ConPre = %d\n", i, M_Con[i], M_ConEvt[i], M_ConPre[i]);
////	}
////#pragma endregion 生成约束
////#pragma region 子问题约束队列
////	cudaMallocManaged(&S_ConPre, sizeof(int)*SUBCON_SIZE);
////	cudaMallocManaged(&S_ConEvt, sizeof(int3)*SUBCON_SIZE);
////	cudaMallocManaged(&S_Con, sizeof(int3)*SUBCON_SIZE);
////	cudaMallocManaged(&S_VarPre, sizeof(int)*VS_SIZE*MAX_DOM_SIZE*VS_SIZE);
////	cudaMallocManaged(&S_Var, sizeof(int3)*VS_SIZE*MAX_DOM_SIZE*VS_SIZE);
////
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		const XVar* v = xm->vars[i];
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			for (int k = 0; k < CS_SIZE; ++k)
////			{
////				// 子问题(i, j) k为约束id
////				const int idx = (i*MAX_DOM_SIZE + j)*CS_SIZE + k;
////				//i*xm->feature.max_dom_size*xm->feature.cs_size + j*xm->feature.cs_size + k;
////
////				S_Con[idx].x = i;
////				S_Con[idx].y = j;
////				S_Con[idx].z = k;
////
////				S_ConEvt[idx].x = i;
////				S_ConEvt[idx].y = j;
////				S_ConEvt[idx].z = k;
////
////				S_ConPre[idx] = 1;
////				//printf("S_Con = (%d, %d, %d), S_ConEvt = (%d, %d, %d), pre = %d\n", S_Con[idx].x, S_Con[idx].y, S_Con[idx].z, S_ConEvt[idx].x, S_ConEvt[idx].y, S_ConEvt[idx].z, S_ConPre[idx]);
////			}
////
////			for (int k = 0; k < VS_SIZE; ++k)
////			{
////				//子问题(i, j) k为变量id
////				const int idx = (i*MAX_DOM_SIZE + j)*VS_SIZE + k;
////				S_Var[idx].x = i;
////				S_Var[idx].y = j;
////				S_Var[idx].z = k;
////
////				S_VarPre[idx] = 1;
////
////				//printf("S_Var = (%d, %d, %d), S_VarPre = %d\n", S_Var[idx].x, S_Var[idx].y, S_Var[idx].z, S_VarPre[idx]);
////			}
////		}
////	}
////#pragma endregion 子问题约束队列
////
////#pragma region 程序运行规格
////	//获得主问题压缩的BLOCK数
////	MCC_BLOCK = GetTopNum(CS_SIZE, num_threads);
////	MCC_BCount.resize(MCC_BLOCK, 0);
////	MCC_BOffset.resize(MCC_BLOCK, 0);
////	MCC_BlocksCount = thrust::raw_pointer_cast(MCC_BCount.data());
////	MCC_BlocksOffset = thrust::raw_pointer_cast(MCC_BOffset.data());
////#pragma endregion
////
////}
////
////__global__ void ConCheckMain(int* ConEvt, int* btSp, int2* scp)
////{
////	const int c_id = blockIdx.x;
////	//获取约束在bitSup的开始索引
////	const int start_idx = GetBitSupIndexById(c_id);
////	const int2 sp = scp[c_id];
////	extern __shared__ int2[];
////}
////
////void ConstraintCheckMain()
////{
////	//num_threads最好可变
////	ConCheckMain << <CS_SIZE, num_threads >> >();
////}
////
////float SACGPU()
////{
////	//1. 在主问题上执行AC
////	//1.1. 流压缩
////	CompactQueueMain();
////	//1.2. 约束检查
////	ConstraintCheckMain();
////}
////
////void DelGPUModel()
////{
////	cudaFree(scope);
////	cudaFree(bitDom);
////	cudaFree(M_VarPre);
////	cudaFree(bitSubDom);
////	cudaFree(bitSup);
////	cudaFree(M_Con);
////	cudaFree(M_ConEvt);
////	cudaFree(M_ConPre);
////	cudaFree(S_ConPre);
////	cudaFree(S_ConEvt);
////	cudaFree(S_Con);
////	cudaFree(S_Var);
////	cudaFree(S_VarPre);
////}
////
////
